#include "hip/hip_runtime.h"
#include "gpuErrchk.cu"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <nvfunctional>

#define SEARCH_NOT_FOUND 0
#define ADDRESS_LANE 32
#define VALID_KEY_MASK 0x7fffffff
#define DELETED_KEY 0

const long long unsigned EMPTY = 0;
const long long unsigned EMPTY_PAIR = 0;
const long long unsigned EMPTY_POINTER = 0;
#define BASE_SLAB 1

struct Slab {
  unsigned long long *keyValue;
  unsigned long long *next;
};

volatile Slab **slabs = NULL;
__managed__ unsigned num_of_buckets = 0;

__host__ __device__ unsigned hash(unsigned src_key);

__forceinline__ __device__ unsigned long long
ReadSlab(const unsigned long long &next, const unsigned &src_bucket,
         const unsigned laneId, volatile Slab **slabs,
         unsigned num_of_buckets) {
  if (src_bucket >= num_of_buckets) {
    printf("Error\n");
  }
  if (laneId != 31) {
    return slabs[src_bucket][next - 1].keyValue[laneId];
  }
  return *slabs[src_bucket][next - 1].next;
}

__forceinline__ __device__ unsigned long long *
SlabAddress(const unsigned long long &next, const unsigned &src_bucket,
            const unsigned laneId, volatile Slab **slabs,
            unsigned num_of_buckets) {
  if (src_bucket >= num_of_buckets) {
    printf("Error\n");
  }
  if (laneId != 31) {
    // printf("Got addr %p src_bucket %d\n", slabs[src_bucket][next -
    // 1].keyValue + laneId, src_bucket);

    return (slabs[src_bucket][next - 1].keyValue + laneId);
  } else {
    // printf("Got addr for next\n");
    return slabs[src_bucket][next - 1].next;
  }
}

__forceinline__ __device__ unsigned long long warp_allocate() {
  printf("Didn't implement\n");
  return 0;
}
__forceinline__ __device__ unsigned long long deallocate(unsigned long long l) {
  printf("Didn't implement\n");
  return 0;
}

__forceinline__ __device__ void warp_operation(
    bool *__restrict__ is_active, const unsigned *__restrict__ myKey,
    unsigned *__restrict__ myValue,
    const nvstd::function<void(bool *__restrict__, const unsigned *__restrict__,
                               unsigned *__restrict__, unsigned &, unsigned &,
                               unsigned &, unsigned long long &,
                               unsigned long long &, volatile Slab **,
                               unsigned)> &operation,
    volatile Slab **slabs, unsigned num_of_buckets) {
  const int tid = threadIdx.x + blockDim.x * blockIdx.x;
  const unsigned laneId = threadIdx.x & 0x1F;
  unsigned long long next = BASE_SLAB;
  unsigned work_queue = __ballot_sync(~0, is_active[tid]);

  unsigned last_work_queue = work_queue;

  while (work_queue != 0) {
    next = (work_queue != last_work_queue) ? (BASE_SLAB) : next;
    unsigned src_lane = __ffs(work_queue);
    unsigned src_key = __shfl_sync(~0, myKey[tid], src_lane - 1);
    unsigned src_bucket = hash(src_key);
    // if (laneId == 0)
    //  printf("src_lane %d from %d: %d -> %d\n", src_lane, work_queue, src_key,
    //  src_bucket);
    unsigned long long read_data =
        ReadSlab(next, src_bucket, laneId, slabs, num_of_buckets);

    operation(is_active, myKey, myValue, src_lane, src_key, src_bucket,
              read_data, next, slabs, num_of_buckets);
    last_work_queue = work_queue;
    bool activity = is_active[tid];

    work_queue = __ballot_sync(~0, activity);
  }
}

__forceinline__ __device__ void
warp_search(bool *__restrict__ is_active, const unsigned *__restrict__ myKey,
            unsigned *__restrict__ myValue, unsigned &src_lane,
            unsigned &src_key, unsigned &src_bucket,
            unsigned long long &read_data, unsigned long long &next,
            volatile Slab **slabs, unsigned num_of_buckets) {
  const int tid = threadIdx.x + blockDim.x * blockIdx.x;
  const unsigned laneId = threadIdx.x % 32;
  unsigned key = (unsigned)((read_data >> 32) & 0xffffffff);
  unsigned found_lane =
      __ffs(__ballot_sync(~0, key == src_key) & VALID_KEY_MASK);

  if (found_lane != 0) {
    unsigned long long found_value = __shfl_sync(~0, read_data, found_lane - 1);
    if (laneId == src_lane - 1) {
      myValue[tid] = found_value & 0xffffffff;
      is_active[tid] = false;
    }
  } else {
    unsigned long long next_ptr = __shfl_sync(~0, read_data, ADDRESS_LANE - 1);
    if (next_ptr == 0) {
      if (laneId == src_lane - 1) {
        myValue[tid] = SEARCH_NOT_FOUND;
        is_active[tid] = false;
      }
    } else {
      next = next_ptr;
    }
  }
}

__forceinline__ __device__ void
warp_delete(bool *__restrict__ is_active, const unsigned *__restrict__ myKey,
            unsigned *__restrict__ myValue, unsigned &src_lane,
            unsigned &src_key, unsigned &src_bucket,
            unsigned long long &read_data, unsigned long long &next,
            volatile Slab **slabs, unsigned num_of_buckets) {
  const int tid = threadIdx.x + blockDim.x * blockIdx.x;
  const unsigned laneId = threadIdx.x % 32;
  unsigned key = (unsigned)((read_data >> 32) & 0xffffffff);
  unsigned dest_lane = __ffs(__ballot_sync(VALID_KEY_MASK, key == src_key));
  if (dest_lane != 0) {
    if (src_lane - 1 == laneId) {
      *(SlabAddress(next, src_bucket, dest_lane - 1, slabs, num_of_buckets)) =
          DELETED_KEY;
      is_active[tid] = false;
    }
  } else {
    unsigned long long next_ptr = __shfl_sync(~0, read_data, ADDRESS_LANE - 1);
    if (next_ptr == 0) {
      is_active[tid] = false;
    } else {
      next = next_ptr;
    }
  }
}

__forceinline__ __device__ void
warp_replace(bool *__restrict__ is_active, const unsigned *__restrict__ myKey,
             unsigned *__restrict__ myValue, unsigned &src_lane,
             unsigned &src_key, unsigned &src_bucket,
             unsigned long long &read_data, unsigned long long &next,
             volatile Slab **slabs, unsigned num_of_buckets) {
  const int tid = threadIdx.x + blockDim.x * blockIdx.x;
  const unsigned laneId = threadIdx.x % 32;
  unsigned key = (unsigned)((read_data >> 32) & 0xffffffff);
  bool to_share = (key == EMPTY || key == src_key);
  int masked_ballot = __ballot_sync(~0, to_share) & VALID_KEY_MASK;
  unsigned dest_lane = (unsigned)__ffs(masked_ballot);

  if (dest_lane != 0) {
    if (src_lane - 1 == laneId) {
      unsigned long long key = (unsigned long long)myKey[tid];
      unsigned long long value = (unsigned long long)myValue[tid];
      unsigned long long newPair = (key << 32) | value;
      unsigned long long *addr =
          SlabAddress(next, src_bucket, dest_lane - 1, slabs, num_of_buckets);
      unsigned long long old_pair = atomicCAS(addr, 0, newPair);
      if (old_pair == 0) {
        // printf("%d inserted\n", tid);
        is_active[tid] = false;
        __threadfence();
      } else if ((unsigned)((old_pair >> 32) & 0xffffffff) == key) {
        is_active[tid] = false;
        __threadfence();
        // printf("%d %d tried to insert but got %lld\n", tid, laneId,
        // ((old_pair >> 32) & 0xffffffff));
      }
    }
  } else {
    unsigned long long next_ptr = __shfl_sync(~0, read_data, ADDRESS_LANE - 1);
    if (next_ptr == 0) {
      unsigned long long new_slab_ptr = warp_allocate();
      if (laneId == ADDRESS_LANE) {
        unsigned long long temp = 0;
        temp = atomicCAS(
            SlabAddress(next, src_bucket, ADDRESS_LANE, slabs, num_of_buckets),
            EMPTY_POINTER, new_slab_ptr);
        if (temp != EMPTY_POINTER) {
          deallocate(new_slab_ptr);
        }
      }
    } else {
      next = next_ptr;
    }
  }
}

__forceinline__ __host__ __device__ unsigned hash(unsigned src_key) {
  return src_key % num_of_buckets;
}

void setUp(unsigned size, unsigned numberOfSlabsPerBucket) {
  num_of_buckets = size;
  gpuErrchk(hipMallocManaged(&slabs, sizeof(Slab *) * size));
  for (int i = 0; i < size; i++) {

    gpuErrchk(
        hipMallocManaged(&(slabs[i]), sizeof(Slab) * numberOfSlabsPerBucket));
    for (int k = 0; k < numberOfSlabsPerBucket; k++) {
      gpuErrchk(hipMallocManaged(
          (unsigned long long **)&(slabs[i][k].keyValue), sizeof(long) * 31));
      gpuErrchk(hipMallocManaged((unsigned long long **)&(slabs[i][k].next),
                                  sizeof(long)));

      for (int j = 0; j < 31; j++) {
        slabs[i][k].keyValue[j] = 0; // EMPTY_PAIR;
      }
      if (k < numberOfSlabsPerBucket - 1) {
        *slabs[i][k].next = (long)(k + 2);
      } else {
        *slabs[i][k].next = 0; // EMPTY_POINTER;
      }
    }
  }
}